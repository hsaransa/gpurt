#include "hip/hip_runtime.h"
#include "cudavec.h"
#include <hip/hip_math_constants.h>

#ifndef BLOCK_WIDTH
#  define BLOCK_WIDTH 32
#  define BLOCK_HEIGHT 2
#endif

__constant__ int* result;
__constant__ int2*   nodes;
__constant__ float4* aabbs_x;
__constant__ float4* aabbs_y;
__constant__ float4* aabbs_z;
__constant__ float4* vertices;
__constant__ float4* woop_tris;
__constant__ int width, height;
__constant__ float4 matrix0;
__constant__ float4 matrix1;
__constant__ float4 matrix2;
__constant__ float4 matrix3;
__constant__ int2* zorder;

__device__ int warp_counter;

texture<int2, 1, hipReadModeElementType> tex_nodes;
texture<float4, 1, hipReadModeElementType> tex_aabbs_x;
texture<float4, 1, hipReadModeElementType> tex_aabbs_y;
texture<float4, 1, hipReadModeElementType> tex_aabbs_z;
texture<float4, 1, hipReadModeElementType> tex_vertices;
texture<float4, 1, hipReadModeElementType> tex_woop_tris;

struct shared_block
{
    float3 inv_dir;
};

__device__ float3 get_vertex(int i)
{
    float4 vv = vertices[i];
    //float4 vv = tex1Dfetch(tex_vertices, tri_i);//vertices[tri_i];
    return make_float3(vv.x, vv.y, vv.z);
}

extern "C" __global__ void bvh_trace()
{
    //int block = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
    //int thread_idx = block * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    // Init shared memory.

    __shared__ shared_block shared_memory[BLOCK_WIDTH * BLOCK_HEIGHT];
    shared_block* shared = &shared_memory[threadIdx.y * BLOCK_WIDTH + threadIdx.x];

    __shared__ volatile int ray_index[BLOCK_HEIGHT];
    __shared__ volatile int ray_count[BLOCK_HEIGHT];

    ray_index[threadIdx.y] = 0;
    ray_count[threadIdx.y] = 0;

    while (1)
    {
        // Fetch ray index.

#define QUEUE 1

        if (threadIdx.x == 0 && ray_count[threadIdx.y] == 0)
        {
            ray_index[threadIdx.y] = atomicAdd(&warp_counter, 32*QUEUE);
            ray_count[threadIdx.y] = 32*QUEUE;
        }

        int thread_idx = ray_index[threadIdx.y] + threadIdx.x;

        if (threadIdx.x == 0)
        {
            ray_index[threadIdx.y] += 32;
            ray_count[threadIdx.y] -= 32;
        }

        // Init pixel position.

        if (thread_idx >= width*height)
            break;

        int ix = zorder[thread_idx].x;
        int iy = zorder[thread_idx].y;

        int* resp = &result[iy * width + ix];
        *resp = 0x80FF80;

#if 1
        // Calculate view ray.

        float x0 = (ix + 0.5f) / (float)width * 2.f - 1.f;
        float y0 = (iy + 0.5f) / (float)height * 2.f - 1.f;

        float4 clip0 = make_float4(x0, y0, -1.f, 1.f);
        float4 hp0 = make_float4(dot(matrix0, clip0), dot(matrix1, clip0),
                dot(matrix2, clip0), dot(matrix3, clip0));
        float3 p0 = make_float3(hp0.x, hp0.y, hp0.z) * (1.0f / hp0.w);

        float4 clip1 = make_float4(x0, y0,  1.f, 1.f);
        float4 hp1 = make_float4(dot(matrix0, clip1), dot(matrix1, clip1),
                dot(matrix2, clip1), dot(matrix3, clip1));
        float3 p1 = make_float3(hp1.x, hp1.y, hp1.z) * (1.0f / hp1.w);

        float3 orig = p0;
        float3 dir = p1 - p0;

        float3 inv_dir;
        inv_dir.x = dir.x == 0.f ? 1e-32 : 1.f / dir.x;
        inv_dir.y = dir.y == 0.f ? 1e-32 : 1.f / dir.y;
        inv_dir.z = dir.z == 0.f ? 1e-32 : 1.f / dir.z;

        shared->inv_dir.x = inv_dir.x;
        shared->inv_dir.y = inv_dir.y;
        shared->inv_dir.z = inv_dir.z;

        // Trace.

        int stack[64];
        int sp = 0;
        int node_idx = 0;

        stack[63] = (int)resp; // spill result pointer to local stack

        float hit_t = HIP_INF_F;

#define EXIT_NODE 0x66666666

        int debug = 0;
        while (node_idx != EXIT_NODE)
        {
            if (debug++ > 500)
                break;

            if (node_idx >= 0)
            {
                //float3 orig = make_float3(shared[0], shared[1], shared[2]);
                float3 orig_inv_dir = make_float3(-orig.x * shared->inv_dir.x, -orig.y * shared->inv_dir.y, -orig.z * shared->inv_dir.z);
                float tmin0, tmax0, tmin1, tmax1;

                tmin0 = tmin1 = 0.f;
                tmax0 = tmax1 = hit_t;

                {
                    float4 aabb = tex1Dfetch(tex_aabbs_x, node_idx);//aabbs_x[node_idx];
                    //float4 aabb = aabbs_x[node_idx];

                    float a0 = aabb.x * shared->inv_dir.x + orig_inv_dir.x;
                    float a1 = aabb.y * shared->inv_dir.x + orig_inv_dir.x;
                    tmin0 = fmaxf(fminf(a0, a1), tmin0);
                    tmax0 = fminf(fmaxf(a0, a1), tmax0);

                    float b0 = aabb.z * shared->inv_dir.x + orig_inv_dir.x;
                    float b1 = aabb.w * shared->inv_dir.x + orig_inv_dir.x;
                    tmin1 = fmaxf(fminf(b0, b1), tmin1);
                    tmax1 = fminf(fmaxf(b0, b1), tmax1);
                }

                {
                    float4 aabb = tex1Dfetch(tex_aabbs_y, node_idx);//aabbs_x[node_idx];
                    //float4 aabb = aabbs_y[node_idx];

                    float a0 = aabb.x * shared->inv_dir.y + orig_inv_dir.y;
                    float a1 = aabb.y * shared->inv_dir.y + orig_inv_dir.y;
                    tmin0 = fmaxf(tmin0, fminf(a0, a1));
                    tmax0 = fminf(tmax0, fmaxf(a0, a1));

                    float b0 = aabb.z * shared->inv_dir.y + orig_inv_dir.y;
                    float b1 = aabb.w * shared->inv_dir.y + orig_inv_dir.y;
                    tmin1 = fmaxf(tmin1, fminf(b0, b1));
                    tmax1 = fminf(tmax1, fmaxf(b0, b1));
                }

                {
                    float4 aabb = tex1Dfetch(tex_aabbs_z, node_idx);//aabbs_x[node_idx];
                    //float4 aabb = aabbs_z[node_idx];

                    float a0 = aabb.x * shared->inv_dir.z + orig_inv_dir.z;
                    float a1 = aabb.y * shared->inv_dir.z + orig_inv_dir.z;
                    tmin0 = fmaxf(tmin0, fminf(a0, a1));
                    tmax0 = fminf(tmax0, fmaxf(a0, a1));

                    float b0 = aabb.z * shared->inv_dir.z + orig_inv_dir.z;
                    float b1 = aabb.w * shared->inv_dir.z + orig_inv_dir.z;
                    tmin1 = fmaxf(tmin1, fminf(b0, b1));
                    tmax1 = fminf(tmax1, fmaxf(b0, b1));
                }

                int2 n = tex1Dfetch(tex_nodes, node_idx);

                if (tmin0 <= tmax0)
                {
                    if (tmin1 <= tmax1)
                    {
                        if (tmin1 < tmin0)
                        {
                            int t = n.x; n.x = n.y; n.y = t;
                        }
                        stack[sp++] = n.y;
                        node_idx = n.x;
                    }
                    else
                        node_idx = n.x;
                }
                else
                {
                    if (tmin1 <= tmax1)
                    {
                        node_idx = n.y;
                    }
                    else
                    {
                        if (sp)
                            node_idx = stack[--sp];
                        else
                            node_idx = EXIT_NODE;
                    }
                }
            }

            if (node_idx < 0)
            {
                int2 plop = tex1Dfetch(tex_nodes, -node_idx);
                int tri_i = plop.x;
                int tri_end = tri_i + plop.y;

                //float orig = make_float3(shared[0], shared[1], shared[2]) - v0;
                while (tri_i < tri_end)
                {

                    // Woop's triangle intersection wasn't as good.

#if 0
                    float3 dir = make_float3(
                            1.f / shared->inv_dir.x,
                            1.f / shared->inv_dir.y,
                            1.f / shared->inv_dir.z);

                    float4 v0 = woop_tris[tri_i];

                    float Oz = v0.w - dot(orig, xyz(v0));
                    float invDz = 1.f / dot(dir, xyz(v0));
                    float t = Oz * invDz;

                    if (t > 0.f && t < hit_t)
                    {
                        float4 v1 = woop_tris[tri_i+1];
                        float Ox = v1.w + dot(orig, xyz(v1));
                        float Dx = dot(dir, xyz(v1));
                        float u = Ox + t*Dx;
                        if (u >= 0.f)
                        {
                            float4 v2 = woop_tris[tri_i+2];
                            float Oy = v2.w + dot(orig, xyz(v2));
                            float Dy = dot(dir, xyz(v2));
                            float v = Oy + t*Dy;
                            if (v >= 0.f && u + v <= 1.f)
                            {
                                hit_t = t;
                                //hit_u = u;
                                //hit_v = v;
                                int r = t * 1000.f;
                                int g = t * 1500.f;
                                int b = t * 200.f;
                                *(int*)stack[63] = r | g << 8 | b << 16;
                            }
                        }
                    }

#else

                    // Moller-Trumbore triangle intersection

                    float3 v0 = get_vertex(tri_i);
                    float3 v1 = get_vertex(tri_i+1);
                    float3 v2 = get_vertex(tri_i+2);

                    float3 E1 = v1 - v0;
                    float3 E2 = v2 - v0;
                    float3 T = orig - v0;
                    float3 P = cross(dir, E2);

                    float inv_det = 1.f / dot(E1, P);

                    //float3 T = orig - v0;

                    float u = dot(T, P) * inv_det;

                    if (u >= 0.0f && u <= 1.0f)
                    {
                        float3 Q = cross(T, E1);

                        float v = dot(dir, Q) * inv_det;
                        if (v >= 0.0f && u + v <= 1.0f)
                        {
                            float t = dot(E2, Q) * inv_det;
                            if (t >= 0.0f && t < hit_t)
                            {
                                hit_t = t;
                                //hit_u = u;
                                //hit_v = v;

                                // "shading" is done for each intersection to
                                // save registers

                                // Refetch to save registers.
                                v0 = get_vertex(tri_i);
                                v1 = get_vertex(tri_i+1);
                                v2 = get_vertex(tri_i+2);

                                float3 n = cross(v1 - v0, v2 - v0);
                                n = normalize(n);

                                int r = (n.x * 0.5f + 0.5f) * 255.f;
                                int g = (n.y * 0.5f + 0.5f) * 255.f;
                                int b = (n.z * 0.5f + 0.5f) * 255.f;

                                *(int*)stack[63] = r | g << 8 | b << 16;
                            }
                        }
                    }
#endif

                    tri_i += 3;
                }

                if (sp)
                    node_idx = stack[--sp];
                else
                    node_idx = EXIT_NODE;
            }
        }
#endif
    }
}
